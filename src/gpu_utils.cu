#include "gpuzip.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// GPU Memory Management Helper Functions
class GPUMemoryManager {
public:
    static bool allocateMemory(void** ptr, size_t size) {
        hipError_t status = hipMalloc(ptr, size);
        if (status != hipSuccess) {
            std::cerr << "Failed to allocate GPU memory: " 
                      << hipGetErrorString(status) << std::endl;
            return false;
        }
        return true;
    }

    static bool copyToDevice(void* dst, const void* src, size_t size) {
        hipError_t status = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
        if (status != hipSuccess) {
            std::cerr << "Failed to copy data to GPU: " 
                      << hipGetErrorString(status) << std::endl;
            return false;
        }
        return true;
    }

    static bool copyToHost(void* dst, const void* src, size_t size) {
        hipError_t status = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
        if (status != hipSuccess) {
            std::cerr << "Failed to copy data from GPU: " 
                      << hipGetErrorString(status) << std::endl;
            return false;
        }
        return true;
    }

    static void freeMemory(void* ptr) {
        if (ptr != nullptr) {
            hipFree(ptr);
        }
    }
};

// CUDA Error Checking Helper
class CUDAErrorChecker {
public:
    static bool checkLastError(const char* errorMessage) {
        hipError_t status = hipGetLastError();
        if (status != hipSuccess) {
            std::cerr << errorMessage << ": " 
                      << hipGetErrorString(status) << std::endl;
            return false;
        }
        return true;
    }

    static bool checkKernelExecution(const char* kernelName) {
        hipError_t status = hipDeviceSynchronize();
        if (status != hipSuccess) {
            std::cerr << "Kernel execution failed (" << kernelName << "): " 
                      << hipGetErrorString(status) << std::endl;
            return false;
        }
        return true;
    }
};

// GPU Device Properties Helper
class GPUDeviceProperties {
public:
    static bool getOptimalBlockSize(int* blockSize) {
        hipDeviceProp_t prop;
        hipError_t status = hipGetDeviceProperties(&prop, 0);
        if (status != hipSuccess) {
            std::cerr << "Failed to get device properties: " 
                      << hipGetErrorString(status) << std::endl;
            return false;
        }
        
        *blockSize = prop.maxThreadsPerBlock;
        return true;
    }

    static bool getDeviceMemory(size_t* totalMem, size_t* freeMem) {
        hipError_t status = hipMemGetInfo(freeMem, totalMem);
        if (status != hipSuccess) {
            std::cerr << "Failed to get device memory info: " 
                      << hipGetErrorString(status) << std::endl;
            return false;
        }
        return true;
    }
};
