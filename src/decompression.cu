#include "hip/hip_runtime.h"
#include "gpuzip.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace gpuzip {

__global__ void decompressBlock(const uint8_t* input, size_t input_size,
                              uint8_t* output, size_t* output_size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= input_size) return;

    // Read compression flag
    uint32_t match_info;
    memcpy(&match_info, input + idx * sizeof(uint32_t), sizeof(uint32_t));
    
    uint32_t match_length = match_info >> 24;
    uint32_t match_position = match_info & 0x00FFFFFF;
    
    if (match_length >= MIN_MATCH_LENGTH) {
        // Copy matched sequence
        for (uint32_t i = 0; i < match_length; ++i) {
            output[idx + i] = output[match_position + i];
        }
        atomicMax(output_size, idx + match_length);
    } else {
        // Copy literal byte
        output[idx] = input[idx];
        atomicMax(output_size, idx + 1);
    }
}

} // namespace gpuzip

bool decompressFile(const char* input_path, const char* output_path) {
    // Read compressed file
    std::vector<uint8_t> input_data;
    if (!readFile(input_path, input_data)) {
        return false;
    }

    size_t input_size = input_data.size();
    
    // Allocate device memory
    thrust::device_vector<uint8_t> d_input(input_data);
    thrust::device_vector<uint8_t> d_output(input_size * 4); // Conservative estimate
    
    // Calculate grid dimensions
    const int block_size = 256;
    const int num_blocks = (input_size + block_size - 1) / block_size;
    
    // Decompress data
    size_t* d_output_size;
    hipMalloc(&d_output_size, sizeof(size_t));
    hipMemset(d_output_size, 0, sizeof(size_t));
    
    gpuzip::decompressBlock<<<num_blocks, block_size>>>(
        thrust::raw_pointer_cast(d_input.data()),
        input_size,
        thrust::raw_pointer_cast(d_output.data()),
        d_output_size
    );
    
    // Check for kernel errors
    hipError_t cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        std::cerr << "Kernel error: " << hipGetErrorString(cuda_status) << "\n";
        return false;
    }
    
    // Get decompressed size
    size_t output_size;
    hipMemcpy(&output_size, d_output_size, sizeof(size_t), hipMemcpyDeviceToHost);
    hipFree(d_output_size);
    
    // Copy decompressed data back to host
    std::vector<uint8_t> output_data(output_size);
    thrust::copy(d_output.begin(), d_output.begin() + output_size, output_data.begin());
    
    // Write output file
    return writeFile(output_path, output_data);
}
