#include "hip/hip_runtime.h"
#include "gpuzip.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>
#include <cstring>

namespace gpuzip {

__device__ uint32_t hash(const uint8_t* data, size_t size) {
    uint32_t hash = 2166136261u;
    for (size_t i = 0; i < size; ++i) {
        hash ^= data[i];
        hash *= 16777619u;
    }
    return hash;
}

__device__ bool compareSequences(const uint8_t* a, const uint8_t* b, size_t max_len) {
    for (size_t i = 0; i < max_len; ++i) {
        if (a[i] != b[i]) return false;
    }
    return true;
}

__global__ void findMatches(const uint8_t* input, size_t input_size,
                          uint32_t* match_lengths, uint32_t* match_positions) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= input_size) return;

    // Initialize with no match
    match_lengths[idx] = 0;
    match_positions[idx] = 0;

    // Don't look for matches near the end of the input
    if (idx > input_size - MIN_MATCH_LENGTH) return;

    // Compute hash of current position
    uint32_t cur_hash = hash(input + idx, MIN_MATCH_LENGTH);
    
    // Search for matches in the sliding window
    const int window_size = 32768; // 32KB sliding window
    const int start = max(0, idx - window_size);
    
    for (int pos = start; pos < idx; pos++) {
        if (pos + MIN_MATCH_LENGTH > input_size) break;
        
        uint32_t pos_hash = hash(input + pos, MIN_MATCH_LENGTH);
        if (pos_hash == cur_hash && compareSequences(input + idx, input + pos, MIN_MATCH_LENGTH)) {
            // Found a potential match, try to extend it
            size_t len = MIN_MATCH_LENGTH;
            while (idx + len < input_size && pos + len < input_size &&
                   input[idx + len] == input[pos + len] && len < MAX_MATCH_LENGTH) {
                len++;
            }
            
            if (len > match_lengths[idx]) {
                match_lengths[idx] = len;
                match_positions[idx] = pos;
            }
        }
    }
}

__global__ void compressBlock(const uint8_t* input, size_t input_size,
                            const uint32_t* match_lengths,
                            const uint32_t* match_positions,
                            uint8_t* output, size_t* output_size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= input_size) return;

    // Simple byte-by-byte compression for now
    // TODO: Implement advanced entropy coding
    if (match_lengths[idx] >= MIN_MATCH_LENGTH) {
        // Write match information
        uint32_t match_info = (match_lengths[idx] << 24) | match_positions[idx];
        memcpy(output + idx * sizeof(uint32_t), &match_info, sizeof(uint32_t));
    } else {
        // Write literal byte
        output[idx] = input[idx];
    }
}

} // namespace gpuzip

bool compressFile(const char* input_path, const char* output_path, const CompressionParams& params) {
    // Read input file
    std::vector<uint8_t> input_data;
    if (!readFile(input_path, input_data)) {
        return false;
    }

    size_t input_size = input_data.size();
    
    // Allocate device memory
    thrust::device_vector<uint8_t> d_input(input_data);
    thrust::device_vector<uint32_t> d_match_lengths(input_size);
    thrust::device_vector<uint32_t> d_match_positions(input_size);
    thrust::device_vector<uint8_t> d_output(input_size * 2); // Worst case size
    
    // Calculate grid dimensions
    const int block_size = 256;
    const int num_blocks = (input_size + block_size - 1) / block_size;
    
    // Find matches
    gpuzip::findMatches<<<num_blocks, block_size>>>(
        thrust::raw_pointer_cast(d_input.data()),
        input_size,
        thrust::raw_pointer_cast(d_match_lengths.data()),
        thrust::raw_pointer_cast(d_match_positions.data())
    );
    
    // Check for kernel errors
    hipError_t cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        std::cerr << "Kernel error: " << hipGetErrorString(cuda_status) << "\n";
        return false;
    }
    
    // Compress data
    size_t* d_output_size;
    hipMalloc(&d_output_size, sizeof(size_t));
    hipMemset(d_output_size, 0, sizeof(size_t));
    
    gpuzip::compressBlock<<<num_blocks, block_size>>>(
        thrust::raw_pointer_cast(d_input.data()),
        input_size,
        thrust::raw_pointer_cast(d_match_lengths.data()),
        thrust::raw_pointer_cast(d_match_positions.data()),
        thrust::raw_pointer_cast(d_output.data()),
        d_output_size
    );
    
    // Get compressed size
    size_t output_size;
    hipMemcpy(&output_size, d_output_size, sizeof(size_t), hipMemcpyDeviceToHost);
    hipFree(d_output_size);
    
    // Copy compressed data back to host
    std::vector<uint8_t> output_data(output_size);
    thrust::copy(d_output.begin(), d_output.begin() + output_size, output_data.begin());
    
    // Write output file
    return writeFile(output_path, output_data);
}
