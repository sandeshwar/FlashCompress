
#include <hip/hip_runtime.h>
// #include "gpuzip.cuh"
// #include <iostream>
// #include <fstream>
// #include <vector>
// #include <string>
// #include <chrono>

// void printUsage() {
//     std::cout << "Usage: gpuzip [compress|decompress] input_file output_file\n";
// }

// bool readFile(const char* path, std::vector<uint8_t>& data) {
//     std::ifstream file(path, std::ios::binary);
//     if (!file) {
//         std::cerr << "Error: Cannot open input file: " << path << "\n";
//         return false;
//     }
    
//     file.seekg(0, std::ios::end);
//     size_t size = file.tellg();
//     file.seekg(0, std::ios::beg);
    
//     data.resize(size);
//     file.read(reinterpret_cast<char*>(data.data()), size);
//     return true;
// }

// bool writeFile(const char* path, const std::vector<uint8_t>& data) {
//     std::ofstream file(path, std::ios::binary);
//     if (!file) {
//         std::cerr << "Error: Cannot create output file: " << path << "\n";
//         return false;
//     }
    
//     file.write(reinterpret_cast<const char*>(data.data()), data.size());
//     return true;
// }

// int main(int argc, char* argv[]) {
//     if (argc != 4) {
//         printUsage();
//         return 1;
//     }

//     std::string mode = argv[1];
//     const char* input_path = argv[2];
//     const char* output_path = argv[3];

//     // Initialize CUDA
//     cudaError_t cuda_status = cudaSetDevice(0);
//     if (cuda_status != cudaSuccess) {
//         std::cerr << "Error: CUDA device initialization failed: "
//                   << cudaGetErrorString(cuda_status) << "\n";
//         return 1;
//     }

//     // Set compression parameters
//     CompressionParams params{
//         .block_size = BLOCK_SIZE,
//         .dictionary_size = DICTIONARY_SIZE,
//         .compression_level = 9.0f
//     };

//     bool success;
//     auto start_time = std::chrono::high_resolution_clock::now();

//     if (mode == "compress") {
//         success = compressFile(input_path, output_path, params);
//     } else if (mode == "decompress") {
//         success = decompressFile(input_path, output_path);
//     } else {
//         std::cerr << "Error: Invalid mode. Use 'compress' or 'decompress'\n";
//         printUsage();
//         return 1;
//     }

//     auto end_time = std::chrono::high_resolution_clock::now();
//     auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

//     if (success) {
//         std::cout << "Operation completed successfully in " << duration.count() << "ms\n";
//         return 0;
//     } else {
//         std::cerr << "Operation failed\n";
//         return 1;
//     }
// }
